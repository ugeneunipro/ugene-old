
#include <hip/hip_runtime.h>
/**
* UGENE - Integrated Bioinformatics Tools.
* Copyright (C) 2008-2016 UniPro <ugene@unipro.ru>
* http://ugene.unipro.ru
*
* This program is free software; you can redistribute it and/or
* modify it under the terms of the GNU General Public License
* as published by the Free Software Foundation; either version 2
* of the License, or (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program; if not, write to the Free Software
* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
* MA 02110-1301, USA.
*/

typedef int ScoreType;

// CUDA global constants
__constant__ int g_partSeqSize;
__constant__ int g_partsNumber;
__constant__ int g_overlapLength;
__constant__ int g_seqLibLength;
__constant__ int g_queryLength;
__constant__ int g_gapOpen;
__constant__ int g_gapExtension;
__constant__ int g_maxScore;
__constant__ int g_queryPartLength;
__constant__ char g_upSymbolDirectMatrix;
__constant__ char g_leftSymbolDirectMatrix;
__constant__ char g_diagSymbolDirectMatrix;
__constant__ char g_stopSymbolDirectMatrix;

//global function
__global__ void calculateMatrix(const char * seqLib, ScoreType* queryProfile,
                                ScoreType* g_HdataUp, ScoreType* g_HdataRec, ScoreType* g_HdataMax,
                                ScoreType* g_FdataUp,
                                ScoreType* g_directionsUp, ScoreType* g_directionsRec, ScoreType* g_directionsMax,
                                int queryStartPos, int * g_directionsMatrix, int * g_backtraceBegins)
{
    //registers
    int patternPos = threadIdx.x;
    int globalPatternPos =  queryStartPos + patternPos;

    int seqStartPos = blockIdx.x * (g_partSeqSize - g_overlapLength);
    int globalStartPos = blockIdx.x * (g_partSeqSize + 1);
    
    int seqPos = 0;
    int globalPos = 0;
    int diagNum = 0;

    ScoreType substScore = 0;
    ScoreType E = 0;
    ScoreType E_left = 0;
    ScoreType F = 0;
    ScoreType F_up = 0;
    ScoreType H = 0;
    ScoreType H_left = 0;
    ScoreType H_up = 0;
    ScoreType H_upleft = 0;
    ScoreType directionLeft = 0;
    ScoreType directionUp = 0;
    ScoreType directionUpLeft = 0;
    ScoreType direction = 0;

    //dynamic allocation shared memory
    extern __shared__ ScoreType shared_H[];
    ScoreType* shared_E = (ScoreType*)&shared_H[g_queryPartLength + 1];
    ScoreType* shared_direction = (ScoreType*)&shared_E[g_queryPartLength + 1];

    shared_H[patternPos] = 0;
    shared_E[patternPos] = 0;
    shared_direction[patternPos] = 0;

    __syncthreads();

    seqPos = seqStartPos - patternPos;
    globalPos = globalStartPos - patternPos;

    diagNum = g_partSeqSize + g_queryPartLength - 1;
    if (diagNum == 0) {
        return;
    }

    for (int iteration = 0; iteration < diagNum; iteration++) {
        //check boundaries
        bool isActual = seqPos < g_seqLibLength && seqPos >= seqStartPos && seqPos < seqStartPos + g_partSeqSize && globalPatternPos < g_queryLength;
        if (isActual) {
            substScore = queryProfile[seqLib[seqPos] * g_queryLength + globalPatternPos];

            H_left = shared_H[patternPos];
            E_left = shared_E[patternPos];
            directionLeft = shared_direction[patternPos];

            if (patternPos == 0) {

                H_left = g_HdataUp[globalPos];
                E_left = g_FdataUp[globalPos];
                directionLeft = g_directionsUp[globalPos];

                if (globalPos > 0) {
                    H_upleft = g_HdataUp[globalPos - 1];
                    directionUpLeft = g_directionsUp[globalPos - 1];

                } 
                if (queryStartPos == 0 || iteration == 0) {
                    directionUpLeft = seqPos;
                }   
            }
        }
        __syncthreads();

        if (isActual) {
            E = max(E_left - g_gapExtension, H_left - g_gapOpen);
            F = max(F_up - g_gapExtension, H_up - g_gapOpen);

            // Compute H
            H = max(0, E);
            H = max(H, F);
            H = max(H, H_upleft + substScore);

            //Collect best results
            if (g_HdataMax[globalPos] <= H_upleft + substScore) {
                g_HdataMax[globalPos] = H_upleft + substScore;
                g_directionsMax[globalPos] = directionUpLeft;

                if (NULL != g_directionsMatrix && NULL != g_backtraceBegins) {
                    g_backtraceBegins[globalPos * 2] = globalPatternPos;
                    g_backtraceBegins[globalPos * 2 + 1] = seqPos;
                }
            }

            //chose direction
            char directionForMatrix = g_stopSymbolDirectMatrix;

            if (H == 0) {
                direction = seqPos + 1;
            }
            else if (H == H_upleft + substScore) {
                direction = directionUpLeft;
                directionForMatrix = g_diagSymbolDirectMatrix;
            }
            else if (H == F) {
                direction = directionUp;
                directionForMatrix = g_leftSymbolDirectMatrix;
            }
            else {
                direction = directionLeft;
                directionForMatrix = g_upSymbolDirectMatrix;
            }                    
                
            shared_E[patternPos + 1] = E;
            shared_H[patternPos + 1] = H;
            shared_direction[patternPos + 1] = direction;

            H_upleft = H_left;
            H_up = H;
            F_up = F;

            directionUp = direction;
            directionUpLeft = directionLeft;

            if(0 != g_directionsMatrix) {
                g_directionsMatrix[g_seqLibLength * globalPatternPos + seqPos] = (int)directionForMatrix;
            }


            //if this last iteration then start prepare next
            if (patternPos == (g_queryPartLength - 1)) {
                g_HdataRec[globalPos] = H;
                g_FdataUp[globalPos] = E;
                g_directionsRec[globalPos]  = direction;
            }
        }
        __syncthreads();

        seqPos++;       // = seqStartPos + iteration - patternPos;
        globalPos++;    // = globalStartPos + iteration - patternPos;
    }
}

void calculateMatrix_wrap(int blockSize, int threadNum, const char * seqLib, ScoreType* queryProfile,                           
                          ScoreType* g_HdataUp, ScoreType* g_HdataRec, ScoreType* g_HdataMax,
                          ScoreType* g_FdataUp,
                          ScoreType* g_directionsUp, ScoreType* g_directionsRec, ScoreType* g_directionsMax,
                          int iteration, int * g_directionsMatrix, int * g_backtraceBegins)
{
    size_t sh_mem_size = sizeof(ScoreType) * (threadNum + 1) * 3;
    calculateMatrix<<<blockSize, threadNum, sh_mem_size>>>(seqLib,
        queryProfile, g_HdataUp, 
        g_HdataRec, g_HdataMax, g_FdataUp,
        g_directionsUp, g_directionsRec, g_directionsMax, iteration,
        g_directionsMatrix, g_backtraceBegins);
}

void setConstants(int partSeqSize,
                  int partsNumber,
                  int overlapLength,
                  int seqLibLength,
                  int queryLength,
                  int gapOpen,
                  int gapExtension,
                  int maxScore,
                  int queryPartLength,
                  char upSymbolDirectMatrix,
                  char leftSymbolDirectMatrix,
                  char diagSymbolDirectMatrix,
                  char stopSymbolDirectMatrix) {
    hipMemcpyToSymbol(HIP_SYMBOL(g_partSeqSize),               &partSeqSize,               sizeof(partSeqSize));
    hipMemcpyToSymbol(HIP_SYMBOL(g_partsNumber),               &partsNumber,               sizeof(partsNumber));
    hipMemcpyToSymbol(HIP_SYMBOL(g_overlapLength),             &overlapLength,             sizeof(overlapLength));
    hipMemcpyToSymbol(HIP_SYMBOL(g_seqLibLength),              &seqLibLength,              sizeof(seqLibLength));
    hipMemcpyToSymbol(HIP_SYMBOL(g_queryLength),               &queryLength,               sizeof(queryLength));
    hipMemcpyToSymbol(HIP_SYMBOL(g_gapOpen),                   &gapOpen,                   sizeof(gapOpen));
    hipMemcpyToSymbol(HIP_SYMBOL(g_gapExtension),              &gapExtension,              sizeof(gapExtension));
    hipMemcpyToSymbol(HIP_SYMBOL(g_maxScore),                  &maxScore,                  sizeof(maxScore));
    hipMemcpyToSymbol(HIP_SYMBOL(g_queryPartLength),           &queryPartLength,           sizeof(queryPartLength));
    hipMemcpyToSymbol(HIP_SYMBOL(g_upSymbolDirectMatrix),      &upSymbolDirectMatrix,      sizeof(upSymbolDirectMatrix));
    hipMemcpyToSymbol(HIP_SYMBOL(g_leftSymbolDirectMatrix),    &leftSymbolDirectMatrix,    sizeof(leftSymbolDirectMatrix));
    hipMemcpyToSymbol(HIP_SYMBOL(g_diagSymbolDirectMatrix),    &diagSymbolDirectMatrix,    sizeof(diagSymbolDirectMatrix));
    hipMemcpyToSymbol(HIP_SYMBOL(g_stopSymbolDirectMatrix),    &stopSymbolDirectMatrix,    sizeof(stopSymbolDirectMatrix));
}
