
#include <hip/hip_runtime.h>
//#ifdef SW2_BUILD_WITH_CUDA

typedef int ScoreType;

// CUDA global constants
__constant__ int partSeqSize, partsNumber, overlapLength, seqLibLength, queryLength;
__constant__ int gapOpen, gapExtension, maxScore, queryPartLength;


//global function
__global__ void calculateMatrix(const char * seqLib, ScoreType* queryProfile,
								ScoreType* g_HdataUp, ScoreType* g_HdataRec, ScoreType* g_HdataMax,
								ScoreType* g_FdataUp,
								ScoreType* g_directionsUp, ScoreType* g_directionsRec, ScoreType* g_directionsMax,
								int queryStartPos) 
{

	//registers
	int patternPos = threadIdx.x;	
	int globalPatternPos =  queryStartPos + patternPos;

	int seqStartPos = blockIdx.x * (partSeqSize - overlapLength);
	int globalStartPos = blockIdx.x * (partSeqSize + 1);
	
	int seqPos = 0, globalPos = 0, diagNum = 0;
	ScoreType substScore = 0;	
	ScoreType E = 0, E_left = 0, F = 0, F_up = 0, H = 0,
		H_left = 0, H_up = 0, H_upleft = 0, E_left_init = 0, 
		H_left_init = 0, directionLeft = 0, directionUp = 0, 
		directionUpLeft = 0, direction = 0, directionInit = 0,
		maxScore = 0;



	//dynamic allocation shared memory
	extern __shared__ ScoreType shared_H[];
	ScoreType* shared_E = (ScoreType*)&shared_H[queryPartLength + 1];
	ScoreType* shared_direction = (ScoreType*)&shared_E[queryPartLength + 1];

	shared_H[patternPos] = 0;
	shared_E[patternPos] = 0;
	shared_direction[patternPos] = 0;

	__syncthreads();	

	seqPos = seqStartPos - patternPos;
	globalPos = globalStartPos - patternPos;

	diagNum = (partSeqSize + queryPartLength - 1);
	for (int iteration = 0; iteration < diagNum; iteration++) {

		//check boundaries
		bool isActual = seqPos < seqLibLength && seqPos >= seqStartPos && seqPos < seqStartPos + partSeqSize && globalPatternPos < queryLength;
		if (isActual) {
			substScore = queryProfile[seqLib[seqPos] * queryLength + globalPatternPos];

			H_left = shared_H[patternPos];
			E_left = shared_E[patternPos];
			directionLeft = shared_direction[patternPos];

			if (patternPos == 0) {

				H_left = g_HdataUp[globalPos];
				E_left = g_FdataUp[globalPos];
				directionLeft = g_directionsUp[globalPos];

				if (globalPos > 0) {
					H_upleft = g_HdataUp[globalPos - 1];
					directionUpLeft = g_directionsUp[globalPos - 1];

				} 
				if (queryStartPos == 0 || iteration == 0) {
					directionUpLeft = seqPos;				
				}	
			}
		}
		__syncthreads();

		if (isActual) {
			E = max(E_left - gapExtension, H_left - gapOpen);
			F = max(F_up - gapExtension, H_up - gapOpen);

			// Compute H
			H = max(0, E);
			H = max(H, F);
			H = max(H, H_upleft + substScore);	

			//chose direction
			if (H == 0) {		 
				direction = seqPos + 1;				
			}
			else if (H == E) {
				direction = directionLeft;
			}	
			else if (H == F) {
				direction = directionUp;
			}	
			//(H == H_upleft + substScore)
			else {
				direction = directionUpLeft;
			}		
				
			shared_E[patternPos + 1] = E;
			shared_H[patternPos + 1] = H;
			shared_direction[patternPos + 1] = direction;

			H_upleft = H_left;
			H_up = H;
			F_up = F;

			directionUp = direction;			
			directionUpLeft = directionLeft;

			//collect best result
			maxScore = max(H, g_HdataMax[globalPos]);
			if (maxScore == H) {
				g_HdataMax[globalPos] = maxScore;
				g_directionsMax[globalPos] = direction;
			}

			//if this last iteration then start prepare next
			if (patternPos == (queryPartLength - 1)) {
				g_HdataRec[globalPos] = H;
				g_FdataUp[globalPos] = E;			
				g_directionsRec[globalPos]  = direction;
			}
		}
		__syncthreads();

		seqPos++;// = seqStartPos + iteration - patternPos;
		globalPos++;// = globalStartPos + iteration - patternPos;

	}

}


void calculateMatrix_wrap(int blockSize, int threadNum, const char * seqLib, ScoreType* queryProfile,                           
						  ScoreType* g_HdataUp, ScoreType* g_HdataRec, ScoreType* g_HdataMax,
						  ScoreType* g_FdataUp,
						  ScoreType* g_directionsUp, ScoreType* g_directionsRec, ScoreType* g_directionsMax,
						  int iteration) 
{
	size_t sh_mem_size = sizeof(ScoreType) * (threadNum + 1) * 3;
	calculateMatrix<<<blockSize, threadNum, sh_mem_size>>>(seqLib,
		queryProfile, g_HdataUp, 
		g_HdataRec, g_HdataMax, g_FdataUp,                                   
		g_directionsUp, g_directionsRec, g_directionsMax, iteration);
}

void setConstants(int partSeqSize, int partsNumber, int overlapLength, int seqLibLength, int queryLength, int gapOpen, int gapExtension, int maxScore, int queryPartLength) {
	hipMemcpyToSymbol(HIP_SYMBOL("partSeqSize"),    &partSeqSize,    sizeof(partSeqSize));
	hipMemcpyToSymbol(HIP_SYMBOL("partsNumber"),    &partsNumber,    sizeof(partsNumber));
	hipMemcpyToSymbol(HIP_SYMBOL("overlapLength"),    &overlapLength,    sizeof(overlapLength));
	hipMemcpyToSymbol(HIP_SYMBOL("seqLibLength"),    &seqLibLength,    sizeof(seqLibLength));
	hipMemcpyToSymbol(HIP_SYMBOL("queryLength"),    &queryLength,    sizeof(queryLength));
	hipMemcpyToSymbol(HIP_SYMBOL("gapOpen"),    &gapOpen,    sizeof(gapOpen));
	hipMemcpyToSymbol(HIP_SYMBOL("gapExtension"),    &gapExtension,    sizeof(gapExtension));
	hipMemcpyToSymbol(HIP_SYMBOL("maxScore"),    &maxScore,    sizeof(maxScore));
	hipMemcpyToSymbol(HIP_SYMBOL("queryPartLength"),    &queryPartLength,    sizeof(queryPartLength));
}

//#endif //SW2_BUILD_WITH_CUDA
