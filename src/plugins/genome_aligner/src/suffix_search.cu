#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "cuPrintf.cu"

typedef unsigned long long BMType;
typedef unsigned int SAType;

__global__ void binarySearch(BMType array[], int arraySize, 
            BMType query[], int querySize) {

    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    int left = 0;
    int right = arraySize - 1;
    const BMType target = query[threadId];

    while (left <= right)
    {
        int mid = (left + right) >> 1;
        BMType midValue = array[mid];
        if(midValue > target) {
            right = mid - 1;
        } else if(midValue < target) {
            left = mid + 1;
        } else {
            for (mid = mid - 1; mid >= 0; --mid ) {
                if (array[mid] != target) {
                    break;
                }
            }
            query[threadId] = mid + 1;
            return;
        }
    }

    query[threadId] = -1;

}

extern "C" void cudaBinarySearch(BMType* array, int arraySize, BMType* query, int querySize )
{
    printf("Starting binary search...\n");
    binarySearch <<<querySize / 256 + 1, 256>>> (array, arraySize, query, querySize);
    hipDeviceSynchronize();
    printf("Binary search is finished...\n");
}


struct ShortReads 
{
    char* data;
    int* offsets;
    int* sizes;
    int count;
};

struct BMArray
{
    BMType* data;
    int size;
};

struct SArray
{
    SAType* data;
    int size;
};

struct AlgSettings
{
    BMType bitFilter;
    int refStartPos;
    int loadedPartSize;
    int CMAX;
    int w;
};


__device__ int lowerBound(BMType* bmArray, int bmArraySize, BMType target, BMType bitFilter) {
    int left = 0;
    int right = bmArraySize - 1;
    int result = -1;

    while (left <= right)
    {
        int mid = (left + right) >> 1;
        long long rc = (bmArray[mid]&bitFilter) - (target&bitFilter);
        if(rc > 0) {
            right = mid - 1;
        } else if(rc < 0) {
            left = mid + 1;
        } else {
            for (mid = mid - 1; mid >= 0; --mid ) {
                if ((bmArray[mid]&bitFilter) != (target&bitFilter)) {
                    break;
                }
            }
            result = mid + 1;
            break;
        }
    }

    return result;

}

__device__ BMType calcBitValue(char* readsData, int readOffset, int length, int charsInMask, SAType* bitTable)
{
    const int bitCharLen = 2;
    BMType bitValue = 0;
    //cuPrintf("L = %d, charsInMask = %d\n",length,charsInMask);
    //cuPrintf("bw=%ullx\n", bitValue);
    for (int i = 0; i < length; i++) {
        char c = readsData[readOffset+i];
        //cuPrintf("C=%c\n",c);
        //cuPrintf("bw=%llx\n", bitValue);
        bitValue = (bitValue << bitCharLen) | bitTable[c  - 65];
    }

    bitValue <<= 2*charsInMask - bitCharLen*length;

    return bitValue;    
}

__device__ void initBitTable(SAType* bitTable)
{
    bitTable['A' - 65] = 0x0; //00
    bitTable['C' - 65] = 0x1; //01
    bitTable['T' - 65] = 0x2; //10
    bitTable['G' - 65] = 0x3; //11
}


__device__ bool compare(const char *sourceSeq, const char *querySeq, int startPos, int w, int& c, int CMAX, int length) 
{
    // forward collect
    for (int i=startPos+w; i<length && c <= CMAX; i++) {
        char c1 = querySeq[i];
        char c2 = sourceSeq[i];
        //cuPrintf("F: %c %c pos=%d\n",c1,c2,i);
        c += (c1 == c2) ? 0 : 1;
    }
    if (c>CMAX) {
        return false;
    }
    // backward collect
    for (int i=startPos-1; i>=0 && c <= CMAX; i--) {
        char c1 = querySeq[i];
        char c2 = sourceSeq[i];
        c += (c1 == c2) ? 0 : 1;
        //cuPrintf("B: %c %c pos=%d\n",c1,c2,i);
    }
    if (c <= CMAX) {
        return true;
    }
    return false;
}


__global__ void alignReadsKernel(ShortReads reads,
                               char* refSeq,
                               int refSeqSize,
                               SArray sArray,
                               BMArray bmArray,
                               AlgSettings s,
                               SAType* results)
{

    const int CHARS_IN_MASK = 31;
    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadId >= reads.count) {
        return;
    }
    
    int readOffset = reads.offsets[threadId];
    int readSize = reads.sizes[threadId];

    unsigned int bitTable[32];
    initBitTable(bitTable);
    
    /*
    if (threadId == 0) {
        cuPrintf("Offset is %d\n", readOffset);
        cuPrintf("Read size is: %d\n", readSize);
        cuPrintf("Read first symbol is %c\n", reads.data[0]);
        cuPrintf("First symbol of ref seq is %c\n", refSeq[0]);
        cuPrintf("Read last symbol is %c\n", reads.data[readOffset + readSize - 1]);
    }*/

    int CMAX = s.CMAX;
    int q = readSize / (CMAX + 1);
    SAType result = -1;
    int bestC = 0;
    for (int startPosInRead = 0; startPosInRead < readSize - s.w + 1; startPosInRead += q) {
        int len = readSize - startPosInRead > CHARS_IN_MASK ? CHARS_IN_MASK : readSize - startPosInRead;
        BMType bitValue = calcBitValue(reads.data, readOffset + startPosInRead, len, CHARS_IN_MASK, bitTable);
        SAType bmResult = lowerBound(bmArray.data, bmArray.size, bitValue, s.bitFilter);
        //cuPrintf("BW result is %u\n", bmResult);
        if (bmResult == -1) {
            continue;
        }            
        
        for (SAType k = bmResult; (k < s.loadedPartSize) && (bitValue & s.bitFilter)==(bmArray.data[k] & s.bitFilter); k++) {
            SAType pos = sArray.data[k];
            SAType firstSymbolPos = pos - startPosInRead;
            /*cuPrintf("pos is %d\n", pos);
            cuPrintf("startPosInRead=%d\n", startPosInRead);
            cuPrintf("firstSymbolPos=%d\n", firstSymbolPos);*/
            if (firstSymbolPos == result) {
                continue;
            }
            
            //TODO: check if valid pos in sense of sequence ends...
            /*
            if (!isValidPos( pos + loadedSeqStart, readStartPos, readSize, fisrtSymbol, q)) {
            continue;
            }
            */
            char* refBuff = refSeq + firstSymbolPos;
            char* readBuff = reads.data + readOffset;
            int c = 0;
            if (compare(refBuff, readBuff, startPosInRead, s.w, c, CMAX, readSize)) {
                //cuPrintf("c=%d,CMAX=%d\n",c, CMAX);
                result = firstSymbolPos;
                if (0 == c) {
                    break;
                }
                bestC = c;
                CMAX = bestC - 1;
            }

        }
    }
    //cuPrintf("Result is %d, mismatch rate is %d\n", result,bestC);
    results[2*threadId] = result + s.refStartPos;
    results[2*threadId + 1] = bestC;

}

extern "C" void cudaAlignReads(char* readsData, 
                               int* readSizes,
                               int* readOffsets,
                               int readsNumber,
                               char* refSeq,
                               int refSeqSize,
                               SAType* sArrayData,
                               int sArraySize,
                               BMType* bmArrayData,
                               int bmArraySize,
                               BMType bitFilter,
                               int refStartPos,
                               int loadedPartSize,
                               int CMAX,
                               int w,
                               SAType* results)
{

    printf("Starting aligning reads with CUDA...\n");
    
    ShortReads reads;
    reads.data = readsData;
    reads.sizes = readSizes;
    reads.offsets = readOffsets;
    reads.count = readsNumber;

    BMArray bmArray;
    bmArray.data = bmArrayData;
    bmArray.size = bmArraySize;
    
    SArray sArray;
    sArray.data = sArrayData;
    sArray.size = sArraySize;
    
    AlgSettings settings;
    settings.bitFilter = bitFilter;
    settings.refStartPos = refStartPos;
    settings.loadedPartSize = loadedPartSize;
    
    settings.CMAX = CMAX;
    settings.w = w;
    //cudaPrintfInit(10*1024*1024);

    alignReadsKernel <<<readsNumber / 256 + 1, 256>>> (reads, refSeq, refSeqSize,
        sArray, bmArray, settings, results);
    //cudaPrintfDisplay(stdout, true);
    hipDeviceSynchronize();
    //cudaPrintfEnd();


    printf("Finished aligning reads with CUDA\n");


}
