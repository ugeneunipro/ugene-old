#include "hip/hip_runtime.h"
#include <stdio.h>

typedef unsigned long long NumberType;

__global__ void binarySearch(NumberType array[], int arraySize, 
			NumberType query[], int querySize) {
	
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	
	int left = 0;
	int right = arraySize - 1;
	const NumberType target = query[threadId];
	
	while (left <= right)
	{
		int mid = (left + right) / 2;
		NumberType midValue = array[mid];
		if(midValue > target) {
			right = mid - 1;
		} else if(midValue < target) {
			left = mid + 1;
		} else {
			for (mid = mid - 1; mid >= 0; --mid ) {
				if (array[mid] != target) {
					break;
				}
			}
			query[threadId] = mid + 1;
			return; 
		}
	}
	
	query[threadId] = -1;
		
}

extern "C" void cudaBinarySearch(NumberType* array, int arraySize, NumberType* query, int querySize )
{
	printf("Starting binary search...\n");
	binarySearch <<<querySize / 256 + 1, 256>>> (array, arraySize, query, querySize);	
	hipDeviceSynchronize();
	printf("Binary search is finished...\n");	
}